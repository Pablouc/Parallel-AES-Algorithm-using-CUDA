
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(const int *__restrict a, const int *__restrict b,
                                          int *__restrict c, int N) {
          // Calculate global thread ID
          int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

            // Boundary check
            if (tid < N) c[tid] = a[tid] + b[tid];
}


// Check vector add result and print the operation and its result
void verify_result(std::vector<int> &a, std::vector<int> &b,
                                   std::vector<int> &c) {
          for (int i = 0; i < a.size(); i++) {
                      int expected_result = a[i] + b[i];
                          if (c[i] != expected_result) {
                                        std::cout << "Operation: " << a[i] << " + " << b[i] << " = " << c[i] << " (Expected: " << expected_result << ") [ERROR]\n";
                                                                std::cout << "Id number: " << i ;
                                                                      assert(false); // This will trigger an assertion failure
                                                                          }
                            }
}


int main() {
          hipError_t cudaStatus;
            // Array size of 2^16 (65536 elements)
            constexpr int N = 1 << 8;
              constexpr size_t bytes = sizeof(int) * N;

                // Vectors for holding the host-side (CPU-side) data
                std::vector<int> a;
                  a.reserve(N);
                    std::vector<int> b;
                      b.reserve(N);
                        std::vector<int> c;
                          c.reserve(N);

                            // Initialize random numbers in each array
                            for (int i = 0; i < N; i++) {
                                        a.push_back(rand() % 100);
                                            b.push_back(rand() % 100);
                                              }

                              //std::cout << "Vector A: ";
                                for (int i = 0; i < N; i++) {
                                            //std::cout << a[i] << " ";
                                              }
                                 // std::cout << std::endl;

                                   // std::cout << "Vector B: ";
                                      for (int i = 0; i < N; i++) {
                                     //             std::cout << b[i] << " ";
                                                    }
                                       // std::cout << std::endl;

                                          // Allocate memory on the device
                                          int *d_a, *d_b, *d_c;
                                            cudaStatus = hipMalloc(&d_a, bytes);
                                              hipMalloc(&d_b, bytes);
                                                hipMalloc(&d_c, bytes);

                                                  // Copy data from the host to the device (CPU -> GPU)
                                                  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
                                                    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

                                                      // Threads per CTA (1024)
                                                      int NUM_THREADS = 1 << 10;

                                                        // CTAs per Grid
                                                        // We need to launch at LEAST as many threads as we have elements
                                                        // This equation pads an extra CTA to the grid if N cannot evenly be divided
                                                        // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
                                                        int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

                                                        hipStream_t stream1,stream2;
                                                        hipStreamCreate(&stream1);
                                                        //cudaStreamCreate(&stream2);
                                                        hipEvent_t start, stop;
                                                        hipEventCreate(&start);
                                                        hipEventCreate(&stop);
                                                        hipEventRecord(start);
                                                        // Launch the kernel on the GPU
                                                          // Kernel calls are asynchronous (the CPU program continues execution after
                                                          // call, but no necessarily before the kernel finishes)
                                                          vectorAdd<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(d_a, d_b, d_c, N);
                                                          vectorAdd<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(d_a, d_b, d_c, N);
                                                          //vectorAdd<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(d_a, d_b, d_c, N);

                                                          hipEventRecord(stop);
                                                           hipStreamSynchronize(stream1);
                                                            //cudaStreamSynchronize(stream2);

                                                          float milliseconds = 0;
                                                             hipEventElapsedTime(&milliseconds, start, stop);

                                                            // Copy sum vector from device to host
                                                            // cudaMemcpy is a synchronous operation, and waits for the prior kernel
                                                            // launch to complete (both go to the default stream in this case).
                                                            // Therefore, this cudaMemcpy acts as both a memcpy and synchronization
                                                            // barrier.
                                                            hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

                                                              // Print data on the CPU side after kernel launch
                                                             //std::cout << "Vector C: ";
                                                              for (int i = 0; i < N; i++) {
                                                               //          std::cout << c[i] << " ";
                                                                          }
                                                              // std::cout << std::endl;

                                                                 // Check result for errors
                                                                 verify_result(a, b, c);

                                                                   // Free memory on device
                                                                   hipFree(d_a);
 								       hipFree(d_b);
                                                                       hipFree(d_c);


                                                                      // std::cout << "COMPLETED SUCCESSFULLY\n";

                                                                      // std::cout << "Total execution time:  " << milliseconds << " ms\n";

                                                                 return 0;
}
